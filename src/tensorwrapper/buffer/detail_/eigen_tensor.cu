
#include <hip/hip_runtime.h>
/*
 * Copyright 2025 NWChemEx-Project
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#ifdef ENABLE_CUTENSOR
#include "cutensor_traits.cuh"
#include "eigen_tensor.cuh"
#include <unordered_map>
#include <vector>

namespace tensorwrapper::buffer::detail_ {

// Handle cuda errors
#define HANDLE_CUDA_ERROR(x)                                \
    {                                                       \
        const auto err = x;                                 \
        if(err != cudaSuccess) {                            \
            printf("Error: %s\n", cudaGetErrorString(err)); \
            exit(-1);                                       \
        }                                                   \
    };

// Handle cuTENSOR errors
#define HANDLE_CUTENSOR_ERROR(x)                                \
    {                                                           \
        const auto err = x;                                     \
        if(err != CUTENSOR_STATUS_SUCCESS) {                    \
            printf("Error: %s\n", cutensorGetErrorString(err)); \
            exit(-1);                                           \
        }                                                       \
    };

// Some common typedefs
using mode_vector_t  = std::vector<int>;
using int64_vector_t = std::vector<int64_t>;

// Convert a label into a vector of modes
template<typename LabelType>
mode_vector_t label_to_modes(const LabelType& label) {
    mode_vector_t mode;
    for(const auto& i : label) { mode.push_back(i.data()[0]); }
    return mode;
}

// Query extent information from an input
template<typename InfoType>
int64_vector_t get_extents(const InfoType& info) {
    int64_vector_t extent;
    for(std::size_t i = 0; i < info.rank(); ++i) {
        extent.push_back((int64_t)info.extent(i));
    }
    return extent;
}

// Compute strides in row major
int64_vector_t get_strides(std::size_t N, const int64_vector_t& extent) {
    int64_vector_t strides;
    for(std::size_t i = 0; i < N; ++i) {
        int64_t product = 1;
        for(std::size_t j = N - 1; j > i; --j) product *= extent[j];
        strides.push_back(product);
    }
    return strides;
}

// Perform tensor contraction with cuTENSOR
template<typename TensorType>
void cutensor_contraction(typename TensorType::label_type c_label,
                          typename TensorType::label_type a_label,
                          typename TensorType::label_type b_label,
                          typename TensorType::const_shape_reference c_shape,
                          typename TensorType::const_pimpl_reference A,
                          typename TensorType::const_pimpl_reference B,
                          typename TensorType::eigen_reference C) {
    using element_t    = typename TensorType::element_type;
    using eigen_data_t = typename TensorType::eigen_data_type;

    // GEMM alpha and beta (hardcoded for now)
    element_t alpha = 1.0;
    element_t beta  = 0.0;

    // The modes of the tensors
    mode_vector_t a_modes = label_to_modes(a_label);
    mode_vector_t b_modes = label_to_modes(b_label);
    mode_vector_t c_modes = label_to_modes(c_label);

    // The extents of each tensor
    int64_vector_t a_extents = get_extents(A);
    int64_vector_t b_extents = get_extents(B);
    int64_vector_t c_extents = get_extents(c_shape.as_smooth());

    // The strides of each tensor
    int64_vector_t a_strides = get_strides(A.rank(), a_extents);
    int64_vector_t b_strides = get_strides(B.rank(), b_extents);
    int64_vector_t c_strides = get_strides(c_shape.rank(), c_extents);

    // The size of each tensor
    std::size_t a_size = sizeof(element_t) * A.size();
    std::size_t b_size = sizeof(element_t) * B.size();
    std::size_t c_size = sizeof(element_t) * c_shape.size();

    // Allocate on device
    void *A_d, *B_d, *C_d;
    cudaMalloc((void**)&A_d, a_size);
    cudaMalloc((void**)&B_d, b_size);
    cudaMalloc((void**)&C_d, c_size);

    // Copy to data to device
    HANDLE_CUDA_ERROR(
      cudaMemcpy(A_d, A.get_immutable_data(), a_size, cudaMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(
      cudaMemcpy(B_d, B.get_immutable_data(), b_size, cudaMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(
      cudaMemcpy(C_d, C.data(), c_size, cudaMemcpyHostToDevice));

    // Assert alignment
    const uint32_t kAlignment =
      128; // Alignment of the global-memory device pointers (bytes)
    assert(uintptr_t(A_d) % kAlignment == 0);
    assert(uintptr_t(B_d) % kAlignment == 0);
    assert(uintptr_t(C_d) % kAlignment == 0);

    // cuTENSOR traits
    cutensor_traits<element_t> traits;

    // cuTENSOR handle
    cutensorHandle_t handle;
    HANDLE_CUTENSOR_ERROR(cutensorCreate(&handle));

    // Create Tensor Descriptors
    cutensorTensorDescriptor_t descA;
    HANDLE_CUTENSOR_ERROR(cutensorCreateTensorDescriptor(
      handle, &descA, A.rank(), a_extents.data(), a_strides.data(),
      traits.cutensorDataType, kAlignment));

    cutensorTensorDescriptor_t descB;
    HANDLE_CUTENSOR_ERROR(cutensorCreateTensorDescriptor(
      handle, &descB, B.rank(), b_extents.data(), b_strides.data(),
      traits.cutensorDataType, kAlignment));

    cutensorTensorDescriptor_t descC;
    HANDLE_CUTENSOR_ERROR(cutensorCreateTensorDescriptor(
      handle, &descC, c_shape.rank(), c_extents.data(), c_strides.data(),
      traits.cutensorDataType, kAlignment));

    // Create Contraction Descriptor
    cutensorOperationDescriptor_t desc;
    HANDLE_CUTENSOR_ERROR(cutensorCreateContraction(
      handle, &desc,                               // Base
      descA, a_modes.data(), CUTENSOR_OP_IDENTITY, // A
      descB, b_modes.data(), CUTENSOR_OP_IDENTITY, // B
      descC, c_modes.data(), CUTENSOR_OP_IDENTITY, // C
      descC, c_modes.data(), traits.descCompute    // Result
      ));

    // Ensure that the scalar type is correct.
    cutensorDataType_t scalarType;
    HANDLE_CUTENSOR_ERROR(cutensorOperationDescriptorGetAttribute(
      handle, desc, CUTENSOR_OPERATION_DESCRIPTOR_SCALAR_TYPE,
      (void*)&scalarType, sizeof(scalarType)));
    assert(scalarType == traits.cutensorDataType);

    // Set the algorithm to use
    const cutensorAlgo_t algo = CUTENSOR_ALGO_DEFAULT;
    cutensorPlanPreference_t planPref;
    HANDLE_CUTENSOR_ERROR(cutensorCreatePlanPreference(handle, &planPref, algo,
                                                       CUTENSOR_JIT_MODE_NONE));

    // Query workspace estimate
    uint64_t workspaceSizeEstimate = 0;
    const cutensorWorksizePreference_t workspacePref =
      CUTENSOR_WORKSPACE_DEFAULT;
    HANDLE_CUTENSOR_ERROR(cutensorEstimateWorkspaceSize(
      handle, desc, planPref, workspacePref, &workspaceSizeEstimate));

    // Create Contraction Plan
    cutensorPlan_t plan;
    HANDLE_CUTENSOR_ERROR(
      cutensorCreatePlan(handle, &plan, desc, planPref, workspaceSizeEstimate));

    // Determine workspace size and allocate
    uint64_t actualWorkspaceSize = 0;
    HANDLE_CUTENSOR_ERROR(cutensorPlanGetAttribute(
      handle, plan, CUTENSOR_PLAN_REQUIRED_WORKSPACE, &actualWorkspaceSize,
      sizeof(actualWorkspaceSize)));
    assert(actualWorkspaceSize <= workspaceSizeEstimate);

    void* work = nullptr;
    if(actualWorkspaceSize > 0) {
        HANDLE_CUDA_ERROR(cudaMalloc(&work, actualWorkspaceSize));
        assert(uintptr_t(work) % 128 ==
               0); // workspace must be aligned to 128 byte-boundary
    }

    // Execute
    cudaStream_t stream;
    HANDLE_CUDA_ERROR(cudaStreamCreate(&stream));
    HANDLE_CUTENSOR_ERROR(cutensorContract(handle, plan, (void*)&alpha, A_d,
                                           B_d, (void*)&beta, C_d, C_d, work,
                                           actualWorkspaceSize, stream));

    // Copy Results from Device
    HANDLE_CUDA_ERROR(
      cudaMemcpy(C.data(), C_d, c_size, cudaMemcpyDeviceToHost));

    // Free allocated memory
    HANDLE_CUTENSOR_ERROR(cutensorDestroy(handle));
    HANDLE_CUTENSOR_ERROR(cutensorDestroyPlan(plan));
    HANDLE_CUTENSOR_ERROR(cutensorDestroyOperationDescriptor(desc));
    HANDLE_CUTENSOR_ERROR(cutensorDestroyTensorDescriptor(descA));
    HANDLE_CUTENSOR_ERROR(cutensorDestroyTensorDescriptor(descB));
    HANDLE_CUTENSOR_ERROR(cutensorDestroyTensorDescriptor(descC));
    HANDLE_CUDA_ERROR(cudaStreamDestroy(stream));
    if(A_d) cudaFree(A_d);
    if(B_d) cudaFree(B_d);
    if(C_d) cudaFree(C_d);
    if(work) cudaFree(work);
}

#undef HANDLE_CUTENSOR_ERROR
#undef HANDLE_CUDA_ERROR

// Template instantiations
#define FUNCTION_INSTANTIATE(TYPE, RANK)                         \
    template void cutensor_contraction<EigenTensor<TYPE, RANK>>( \
      typename EigenTensor<TYPE, RANK>::label_type,              \
      typename EigenTensor<TYPE, RANK>::label_type,              \
      typename EigenTensor<TYPE, RANK>::label_type,              \
      typename EigenTensor<TYPE, RANK>::const_shape_reference,   \
      typename EigenTensor<TYPE, RANK>::const_pimpl_reference,   \
      typename EigenTensor<TYPE, RANK>::const_pimpl_reference,   \
      typename EigenTensor<TYPE, RANK>::eigen_reference)

#define DEFINE_CUTENSOR_CONTRACTION(TYPE) \
    FUNCTION_INSTANTIATE(TYPE, 0);        \
    FUNCTION_INSTANTIATE(TYPE, 1);        \
    FUNCTION_INSTANTIATE(TYPE, 2);        \
    FUNCTION_INSTANTIATE(TYPE, 3);        \
    FUNCTION_INSTANTIATE(TYPE, 4);        \
    FUNCTION_INSTANTIATE(TYPE, 5);        \
    FUNCTION_INSTANTIATE(TYPE, 6);        \
    FUNCTION_INSTANTIATE(TYPE, 7);        \
    FUNCTION_INSTANTIATE(TYPE, 8);        \
    FUNCTION_INSTANTIATE(TYPE, 9);        \
    FUNCTION_INSTANTIATE(TYPE, 10)

TW_APPLY_FLOATING_POINT_TYPES(DEFINE_CUTENSOR_CONTRACTION);

#undef DEFINE_CUTENSOR_CONTRACTION
#undef FUNCTION_INSTANTIATE

} // namespace tensorwrapper::buffer::detail_

#endif
