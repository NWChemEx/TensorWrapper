
#include <hip/hip_runtime.h>
/*
 * Copyright 2025 NWChemEx-Project
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#ifdef ENABLE_CUTENSOR
#include "cutensor_traits.cuh"
#include "eigen_tensor.cuh"
#include <unordered_map>
#include <vector>

namespace tensorwrapper::buffer::detail_ {

// Handle cuda errors
#define HANDLE_CUDA_ERROR(x)                                \
    {                                                       \
        const auto err = x;                                 \
        if(err != cudaSuccess) {                            \
            printf("Error: %s\n", cudaGetErrorString(err)); \
            exit(-1);                                       \
        }                                                   \
    };

// Handle cuTENSOR errors
#define HANDLE_CUTENSOR_ERROR(x)                                \
    {                                                           \
        const auto err = x;                                     \
        if(err != CUTENSOR_STATUS_SUCCESS) {                    \
            printf("Error: %s\n", cutensorGetErrorString(err)); \
            exit(-1);                                           \
        }                                                       \
    };

// Some common typedefs
using mode_vector_t  = std::vector<int>;
using int64_vector_t = std::vector<int64_t>;

// Convert a label into a vector of modes
template<typename LabelType>
mode_vector_t label_to_mode_vector(const LabelType& label) {
    mode_vector_t mode;
    for(const auto& i : label) { mode.push_back(i.data()[0]); }
    return mode;
}

// Query extent information from an input
template<typename InfoType>
int64_vector_t get_extents(const InfoType& info) {
    int64_vector_t extent;
    for(std::size_t i = 0; i < info.rank(); ++i) {
        extent.push_back((int64_t)info.extent(i));
    }
    return extent;
}

// Compute strides in row major
int64_vector_t row_major_strides(std::size_t N, const int64_vector_t& extent) {
    int64_vector_t strides;
    for(std::size_t i = 0; i < N; ++i) {
        int64_t product = 1;
        for(std::size_t j = N - 1; j > i; --j) product *= extent[j];
        strides.push_back(product);
    }
    return strides;
}

// Perform tensor contraction with cuTENSOR
template<typename TensorType>
void cutensor_contraction(
  typename TensorType::label_type olabel,
  typename TensorType::label_type llabel,
  typename TensorType::label_type rlabel,
  typename TensorType::const_shape_reference result_shape,
  typename TensorType::const_pimpl_reference lhs,
  typename TensorType::const_pimpl_reference rhs,
  typename TensorType::eigen_reference result) {
    using element_t    = typename TensorType::element_type;
    using eigen_data_t = typename TensorType::eigen_data_type;

    // GEMM alpha and beta (hardcoded for now)
    element_t alpha = 1.0;
    element_t beta  = 0.0;

    // The modes of the tensors
    mode_vector_t lhs_modes    = label_to_mode_vector(llabel);
    mode_vector_t rhs_modes    = label_to_mode_vector(rlabel);
    mode_vector_t output_modes = label_to_mode_vector(olabel);

    // The extents of each tensor
    int64_vector_t lhs_extents    = get_extents(lhs);
    int64_vector_t rhs_extents    = get_extents(rhs);
    int64_vector_t output_extents = get_extents(result_shape.as_smooth());

    // The strides of each tensor
    int64_vector_t lhs_strides = row_major_strides(lhs.rank(), lhs_extents);
    int64_vector_t rhs_strides = row_major_strides(rhs.rank(), rhs_extents);
    int64_vector_t output_strides =
      row_major_strides(result_shape.rank(), output_extents);

    // The size of each tensor
    std::size_t lhs_size    = sizeof(element_t) * lhs.size();
    std::size_t rhs_size    = sizeof(element_t) * rhs.size();
    std::size_t output_size = sizeof(element_t) * result_shape.size();

    // Allocate on device
    void *lhs_d, *rhs_d, *output_d;
    cudaMalloc((void**)&lhs_d, lhs_size);
    cudaMalloc((void**)&rhs_d, rhs_size);
    cudaMalloc((void**)&output_d, output_size);

    // Copy to data to device
    HANDLE_CUDA_ERROR(cudaMemcpy(lhs_d, lhs.get_immutable_data(), lhs_size,
                                 cudaMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(cudaMemcpy(rhs_d, rhs.get_immutable_data(), rhs_size,
                                 cudaMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(
      cudaMemcpy(output_d, result.data(), output_size, cudaMemcpyHostToDevice));

    // Assert alignment
    const uint32_t kAlignment =
      128; // Alignment of the global-memory device pointers (bytes)
    assert(uintptr_t(lhs_d) % kAlignment == 0);
    assert(uintptr_t(rhs_d) % kAlignment == 0);
    assert(uintptr_t(output_d) % kAlignment == 0);

    // cuTENSOR traits
    cutensor_traits<element_t> traits;

    // cuTENSOR handle
    cutensorHandle_t handle;
    HANDLE_CUTENSOR_ERROR(cutensorCreate(&handle));

    // Create Tensor Descriptors
    cutensorTensorDescriptor_t descLHS;
    HANDLE_CUTENSOR_ERROR(cutensorCreateTensorDescriptor(
      handle, &descLHS, lhs.rank(), lhs_extents.data(), lhs_strides.data(),
      traits.cutensorDataType, kAlignment));

    cutensorTensorDescriptor_t descRHS;
    HANDLE_CUTENSOR_ERROR(cutensorCreateTensorDescriptor(
      handle, &descRHS, rhs.rank(), rhs_extents.data(), rhs_strides.data(),
      traits.cutensorDataType, kAlignment));

    cutensorTensorDescriptor_t descOutput;
    HANDLE_CUTENSOR_ERROR(cutensorCreateTensorDescriptor(
      handle, &descOutput, result_shape.rank(), output_extents.data(),
      output_strides.data(), traits.cutensorDataType, kAlignment));

    // Create Contraction Descriptor
    cutensorOperationDescriptor_t desc;
    HANDLE_CUTENSOR_ERROR(cutensorCreateContraction(
      handle, &desc,                                         // Base
      descLHS, lhs_modes.data(), CUTENSOR_OP_IDENTITY,       // A
      descRHS, rhs_modes.data(), CUTENSOR_OP_IDENTITY,       // B
      descOutput, output_modes.data(), CUTENSOR_OP_IDENTITY, // C
      descOutput, output_modes.data(), traits.descCompute    // Output
      ));

    // Optional (but recommended): ensure that the scalar type is correct.
    cutensorDataType_t scalarType;
    HANDLE_CUTENSOR_ERROR(cutensorOperationDescriptorGetAttribute(
      handle, desc, CUTENSOR_OPERATION_DESCRIPTOR_SCALAR_TYPE,
      (void*)&scalarType, sizeof(scalarType)));
    assert(scalarType == traits.cutensorDataType);

    // Set the algorithm to use
    const cutensorAlgo_t algo = CUTENSOR_ALGO_DEFAULT;
    cutensorPlanPreference_t planPref;
    HANDLE_CUTENSOR_ERROR(cutensorCreatePlanPreference(handle, &planPref, algo,
                                                       CUTENSOR_JIT_MODE_NONE));

    // Query workspace estimate
    uint64_t workspaceSizeEstimate = 0;
    const cutensorWorksizePreference_t workspacePref =
      CUTENSOR_WORKSPACE_DEFAULT;
    HANDLE_CUTENSOR_ERROR(cutensorEstimateWorkspaceSize(
      handle, desc, planPref, workspacePref, &workspaceSizeEstimate));

    // Create Contraction Plan
    cutensorPlan_t plan;
    HANDLE_CUTENSOR_ERROR(
      cutensorCreatePlan(handle, &plan, desc, planPref, workspaceSizeEstimate));

    // Optional: Query information about the created plan
    // query actually used workspace
    uint64_t actualWorkspaceSize = 0;
    HANDLE_CUTENSOR_ERROR(cutensorPlanGetAttribute(
      handle, plan, CUTENSOR_PLAN_REQUIRED_WORKSPACE, &actualWorkspaceSize,
      sizeof(actualWorkspaceSize)));
    assert(actualWorkspaceSize <= workspaceSizeEstimate);

    // At this point the user knows exactly how much memory is need by the
    // operation and only the smaller actual workspace needs to be allocated
    void* work = nullptr;
    if(actualWorkspaceSize > 0) {
        HANDLE_CUDA_ERROR(cudaMalloc(&work, actualWorkspaceSize));
        assert(uintptr_t(work) % 128 ==
               0); // workspace must be aligned to 128 byte-boundary
    }

    // Execute
    cudaStream_t stream;
    HANDLE_CUDA_ERROR(cudaStreamCreate(&stream));
    HANDLE_CUTENSOR_ERROR(
      cutensorContract(handle, plan, (void*)&alpha, lhs_d, rhs_d, (void*)&beta,
                       output_d, output_d, work, actualWorkspaceSize, stream));

    // Copy Results from Device
    HANDLE_CUDA_ERROR(
      cudaMemcpy(result.data(), output_d, output_size, cudaMemcpyDeviceToHost));

    // Free allocated memory
    HANDLE_CUTENSOR_ERROR(cutensorDestroy(handle));
    HANDLE_CUTENSOR_ERROR(cutensorDestroyPlan(plan));
    HANDLE_CUTENSOR_ERROR(cutensorDestroyOperationDescriptor(desc));
    HANDLE_CUTENSOR_ERROR(cutensorDestroyTensorDescriptor(descLHS));
    HANDLE_CUTENSOR_ERROR(cutensorDestroyTensorDescriptor(descRHS));
    HANDLE_CUTENSOR_ERROR(cutensorDestroyTensorDescriptor(descOutput));
    HANDLE_CUDA_ERROR(cudaStreamDestroy(stream));
    if(lhs_d) cudaFree(lhs_d);
    if(rhs_d) cudaFree(rhs_d);
    if(output_d) cudaFree(output_d);
    if(work) cudaFree(work);
}

#undef HANDLE_CUTENSOR_ERROR
#undef HANDLE_CUDA_ERROR

// Template instantiations
#define FUNCTION_INSTANTIATE(TYPE, RANK)                         \
    template void cutensor_contraction<EigenTensor<TYPE, RANK>>( \
      typename EigenTensor<TYPE, RANK>::label_type,              \
      typename EigenTensor<TYPE, RANK>::label_type,              \
      typename EigenTensor<TYPE, RANK>::label_type,              \
      typename EigenTensor<TYPE, RANK>::const_shape_reference,   \
      typename EigenTensor<TYPE, RANK>::const_pimpl_reference,   \
      typename EigenTensor<TYPE, RANK>::const_pimpl_reference,   \
      typename EigenTensor<TYPE, RANK>::eigen_reference)

#define DEFINE_CUTENSOR_CONTRACTION(TYPE) \
    FUNCTION_INSTANTIATE(TYPE, 0);        \
    FUNCTION_INSTANTIATE(TYPE, 1);        \
    FUNCTION_INSTANTIATE(TYPE, 2);        \
    FUNCTION_INSTANTIATE(TYPE, 3);        \
    FUNCTION_INSTANTIATE(TYPE, 4);        \
    FUNCTION_INSTANTIATE(TYPE, 5);        \
    FUNCTION_INSTANTIATE(TYPE, 6);        \
    FUNCTION_INSTANTIATE(TYPE, 7);        \
    FUNCTION_INSTANTIATE(TYPE, 8);        \
    FUNCTION_INSTANTIATE(TYPE, 9);        \
    FUNCTION_INSTANTIATE(TYPE, 10)

TW_APPLY_FLOATING_POINT_TYPES(DEFINE_CUTENSOR_CONTRACTION);

#undef DEFINE_CUTENSOR_CONTRACTION
#undef FUNCTION_INSTANTIATE

} // namespace tensorwrapper::buffer::detail_

#endif